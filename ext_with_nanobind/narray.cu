#include "hip/hip_runtime.h"
#include "nanobind/nanobind.h"
#include "nanobind/ndarray.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>

namespace nb = nanobind;

const float data[] = {1, 2, 3, 4, 5, 6, 7, 8};

template<typename T>
__global__ void mul_5(T *data){
  data[0] = data[0] * 5; 
}

void gpu_process(nb::ndarray<uint8_t, nb::shape<nb::any, nb::any>, nb::device::cuda> data) {
  mul_5<<<1,1>>>(data.data());
}

NB_MODULE(narray, m) {
    m.def("inspect", [](nb::ndarray<> a) {
        printf("Array data pointer : %p\n", a.data());
        printf("Array dimension : %zu\n", a.ndim());
        for (size_t i = 0; i < a.ndim(); ++i) {
            printf("Array shape [%zu] : %zu\n", i, a.shape(i));
            printf("Array stride [%zu] : %zu\n", i, a.stride(i));
        }
        printf(
          "Device Id = %u (cpu=%i,gpu = %i)\n",
          a.device_id(),
          int(a.device_type() == nb::device::cpu::value),
          int(a.device_type() == nb::device::cuda::value));
        printf(
          "Array dtype : int16=%i , uint32=%i ,float=%i\n",
          int(a.dtype() == nb::dtype<int16_t>()),
          int(a.dtype() == nb::dtype<uint32_t>()),
          int(a.dtype() == nb::dtype<float>()));
        
        printf("Array num of bytes : %zu\n",a.nbytes());
    });

    m.def(
      "cpu_process", [](nb::ndarray<uint8_t, nb::shape<nb::any, nb::any>, nb::device::cpu> data) {
          data(0, 0) = (uint8_t)data(0, 0) * 5;
      });

    m.def("gpu_process", &gpu_process);

    m.def("ret_numpy", []() {
        size_t shape[2] = {2, 4};
        return nb::ndarray<nb::pytorch, const float, nb::shape<2, nb::any>>(
          data, /* ndim = */ 2, shape);
    });
}